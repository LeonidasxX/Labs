
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

#define N 1024

__global__ void rowSq(unsigned int *a, unsigned int *c, int num_rows){
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if(row < num_rows){
        if(row == 0){
            for (int col = 0; col < num_rows; col++)
            {
                c[row * num_rows + col] = a[row*num_rows + col];
            }
        } else{
            for (int col = 0; col < num_rows; col++)
            {
                c[row * num_rows + col] = pow(a[row*num_rows + col], row+1);
            }
        }
        
    }
}

int main() {
    // int a[3][3], c[3][3];
    unsigned int *dA, *dC;
    unsigned int size;
    int num_rows;

    printf("Enter number of rows: ");
    scanf("%d", &num_rows);

    int a[num_rows][num_rows], c[num_rows][num_rows];

    printf("Enter mat A:\n");
    for (int i = 0; i < num_rows; i++)
    {
        for (int j = 0; j < num_rows; j++)
        {
            scanf("%d", &a[i][j]);
        }
        
    }

    size = num_rows*num_rows*sizeof(int);
    

    hipMalloc((void**) &dA, size);
    hipMalloc((void**) &dC, size);
    hipMemcpy(dA, a, size, hipMemcpyHostToDevice);


    rowSq<<<1, num_rows>>>(dA, dC, num_rows);

    hipMemcpy(c, dC, size, hipMemcpyDeviceToHost);
    
    printf("Resulatant Matrix:\n");
    for (int i = 0; i < num_rows; i++)
    {
        for (int j = 0; j < num_rows; j++)
        {
            printf("%d ", c[i][j]);
        }
        printf("\n");
    }

    hipFree(dA);
    hipFree(dC);
    return 0;
}

