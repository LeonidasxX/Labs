
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

#define N 1024

__global__ void oneS(unsigned int *a, unsigned int *c, int num_rows, int num_cols){
    int row = blockIdx.x * blockDim.x + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(row < num_rows && col < num_cols){
        if(row == 0 || row == num_rows - 1 || col == 0 || col == num_cols-1){
            c[row * num_rows + col] = a[row*num_rows + col];
        } else{
            int n = a[row*num_rows + col];
            if(n > 7){
                int temp[4];
                for (int i = 0; n > 0; i++)
                {
                    temp[i] = n % 2;
                    n = n/2;
                }
                
                for(int i=0; i<4; i++){
                    if(temp[i] == 0) temp[i] = 1;
                    else if(temp[i] == 1) temp[i] = 0;
                }
                int t = 0;
                for(int i=0; i<4; i++){
                    t += temp[i] * pow(10,i);
                }
                c[row * num_rows + col] = t;
            } else{
                int temp[3];
                for (int i = 0; n > 0; i++)
                {
                    temp[i] = n % 2;
                    n = n/2;
                }
                
                for(int i=0; i<3; i++){
                    if(temp[i] == 0) temp[i] = 1;
                    else if(temp[i] == 1) temp[i] = 0;
                }
                int t = 0;
                for(int i=0; i<3; i++){
                    t += temp[i] * pow(10,i);
                }
                c[row * num_rows + col] = t;
            }
            
        }
    }
}

int main() {
    unsigned int *dA, *dC;
    unsigned int size;
    int num_rows;
    int num_cols;

    printf("Enter number of rows: ");
    scanf("%d", &num_rows);
    printf("Enter number of cols: ");
    scanf("%d", &num_cols);

    int a[num_rows][num_cols], c[num_rows][num_cols];

    printf("Enter mat A:\n");
    for (int i = 0; i < num_rows; i++)
    {
        for (int j = 0; j < num_cols; j++)
        {
            scanf("%d", &a[i][j]);
        }
        
    }

    size = num_rows*num_cols*sizeof(int);
    

    hipMalloc((void**) &dA, size);
    hipMalloc((void**) &dC, size);
    hipMemcpy(dA, a, size, hipMemcpyHostToDevice);

    dim3 dimBlock(num_rows,num_cols,1);

    oneS<<<1, (num_rows, dimBlock)>>>(dA, dC, num_rows, num_cols);

    hipMemcpy(c, dC, size, hipMemcpyDeviceToHost);
    
    printf("Resulatant Matrix:\n");
    for (int i = 0; i < num_rows; i++)
    {
        for (int j = 0; j < num_cols; j++)
        {
            printf("%d ", c[i][j]);
        }
        printf("\n");
    }

    hipFree(dA);
    hipFree(dC);
    return 0;
}

