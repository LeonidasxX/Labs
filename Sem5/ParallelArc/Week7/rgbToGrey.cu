#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define CHANNELS 3

__global__ void rgbToGray(unsigned char *inputImage, unsigned char *outputImage, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        int grayOffset = y * width + x;
        int rgbOffset = grayOffset * CHANNELS;

        unsigned char r = inputImage[rgbOffset];
        unsigned char g = inputImage[rgbOffset + 1];
        unsigned char b = inputImage[rgbOffset + 2];

        outputImage[grayOffset] = 0.299f * r + 0.587f * g + 0.114f * b;
    }
}

int main()
{
    // Load RGB image
    unsigned char *h_inputImage;
    int width, height;
    // ... code to load the input image ...

    // Allocate memory in GPU
    unsigned char *d_inputImage;
    unsigned char *d_outputImage;
    size_t imageSize = width * height * CHANNELS * sizeof(unsigned char);
    hipMalloc((void **)&d_inputImage, imageSize);
    hipMalloc((void **)&d_outputImage, width * height * sizeof(unsigned char));

    // Copy input image to GPU
    hipMemcpy(d_inputImage, h_inputImage, imageSize, hipMemcpyHostToDevice);

    // Set up grid and block dimensions
    dim3 dimBlock(16, 16);
    dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (height + dimBlock.y - 1) / dimBlock.y);

    // Launch kernel
    rgbToGray<<<dimGrid, dimBlock>>>(d_inputImage, d_outputImage, width, height);

    // Copy output image from GPU
    unsigned char *h_outputImage = (unsigned char *)malloc(width * height * sizeof(unsigned char));
    hipMemcpy(h_outputImage, d_outputImage, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // ... code to save the output image ...

    // Free memory
    hipFree(d_inputImage);
    hipFree(d_outputImage);
    free(h_outputImage);

    return 0;
}