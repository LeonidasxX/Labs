
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define N 1024

__global__ void addA(int *a, int *b, int *c, int wb, int wa){
    int ridA = threadIdx.x;
    int sum, k;
    if(ridA < wa){
        for(int colIdB = 0; colIdB < wb; colIdB++){
        c[ridA * wa + colIdB] = a[ridA * wa + colIdB] + b[ridA * wa + colIdB];
        }
    }
    
}

__global__ void addB(int *a, int *b, int *c, int wa, int wb){
    int ridB = threadIdx.x;
    int cidB = threadIdx.x;
    if(ridB < wb){
        for (int ridA = 0; ridA < wa; ridA++)
        {
            c[ridA*wa + cidB] = a[ridA*wa + cidB] + b[ridA*wa + cidB];
        }
                
    }
}

__global__ void addC(int *a, int *b, int *c, int wa){
    int ridA = threadIdx.y;
    int cidB = threadIdx.x;
    int wb = blockDim.x;    
    int sum = 0;
    for (int i = 0; i < wa; i++)
    {
        c[ridA*wb+i] = a[ridA*wa+i] + b[i*wb+cidB];
    }
}

int main() {
    int a[3][3], b[3][3], c[3][3];
    int *dA, *dB, *dC;
    unsigned int size, len;

    printf("Enter mat A:\n");
    for (int i = 0; i < 3; i++)
    {
        for (int j = 0; j < 3; j++)
        {
            scanf("%d", &a[i][j]);
        }
        
    }
    printf("Enter mat B:\n");
    for (int i = 0; i < 3; i++)
    {
        for (int j = 0; j < 3; j++)
        {
            scanf("%d", &b[i][j]);
        }
        
    }

    size = 3*3*sizeof(int);
    

    hipMalloc((void**) &dA, size);
    hipMalloc((void**) &dB, size);
    hipMalloc((void**) &dC, size);
    hipMemcpy(dA, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dB, b, size, hipMemcpyHostToDevice);

    dim3 dimBlock(3,3,1);

    // addA<<<1, 3>>>(dA, dB, dC, 3, 3);
    // addB<<<1, 3>>>(dA, dB, dC, 3, 3);
    addC<<<1, dimBlock>>>(dA, dB, dC, 3);

    hipMemcpy(c, dC, size, hipMemcpyDeviceToHost);
    
    printf("Resulatant Matrix:\n");
    for (int i = 0; i < 3; i++)
    {
        for (int j = 0; j < 3; j++)
        {
            printf("%d ", c[i][j]);
        }
        printf("\n");
    }

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    return 0;
}

