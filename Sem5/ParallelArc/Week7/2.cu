
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define N 1024

__global__ void mulA(int *a, int *b, int *c, int wb, int wa){
    int ridA = threadIdx.x;
    int sum, k;
    if(ridA < wa){
        for(int colIdB = 0; colIdB < wb; colIdB++){
        sum = 0;
        for (int k = 0; k < wa; k++)
        {
            sum += a[ridA * wa + k] * b[k * wb + colIdB];
        }
        c[ridA * wa + colIdB] = sum;
    }
    }
    
}

__global__ void mulB(int *a, int *b, int *c, int wa, int wb){
    int cidB = threadIdx.x;
        for (int ridA = 0; ridA < wa; ridA++)
        {
            int sum = 0;
            for (int k = 0; k < wa; k++)
            {
                sum += a[ridA * wa + k] * b[k*wb+cidB];
            }
            c[ridA*wa + cidB] = sum;
        }
                
}

__global__ void mulC(int *a, int *b, int *c, int wa){
    int ridA = threadIdx.y;
    int cidB = threadIdx.x;
    int wb = blockDim.x;    
    int sum = 0;
    for (int i = 0; i < wa; i++)
    {
        sum+= a[ridA*wa+i] * b[i*wb+cidB];
    }
    c[ridA*wb+cidB] = sum;
}

int main() {
    int a[3][3], b[3][3], c[3][3];
    int *dA, *dB, *dC;
    unsigned int size, len;

    printf("Enter mat A:\n");
    for (int i = 0; i < 3; i++)
    {
        for (int j = 0; j < 3; j++)
        {
            scanf("%d", &a[i][j]);
        }
        
    }
    printf("Enter mat B:\n");
    for (int i = 0; i < 3; i++)
    {
        for (int j = 0; j < 3; j++)
        {
            scanf("%d", &b[i][j]);
        }
        
    }

    size = 3*3*sizeof(int);
    

    hipMalloc((void**) &dA, size);
    hipMalloc((void**) &dB, size);
    hipMalloc((void**) &dC, size);
    hipMemcpy(dA, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dB, b, size, hipMemcpyHostToDevice);

    dim3 dimBlock(3,3,1);

    // mulA<<<1, 3>>>(dA, dB, dC, 3, 3);
    // mulB<<<1, 3>>>(dA, dB, dC, 3, 3);
    mulC<<<1, dimBlock>>>(dA, dB, dC, 3);

    hipMemcpy(c, dC, size, hipMemcpyDeviceToHost);
    
    printf("Resulatant Matrix:\n");
    for (int i = 0; i < 3; i++)
    {
        for (int j = 0; j < 3; j++)
        {
            printf("%d ", c[i][j]);
        }
        printf("\n");
    }

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    return 0;
}

