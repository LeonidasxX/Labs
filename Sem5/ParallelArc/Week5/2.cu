#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 5

__global__ void vecAdd(float *a, float *b, float *c) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < N) {
    c[id] = a[id] + b[id];
  }
}

int main() {
  float *h_a = (float *)malloc(N * sizeof(float));
  float *h_b = (float *)malloc(N * sizeof(float));
  float *h_c = (float *)malloc(N * sizeof(float));

  printf("Enter vector A and B: ");
  for (int i = 0; i < N; i++) {
    scanf("%f", &h_a[i]);
    scanf("%f", &h_b[i]);
  }

  float *d_a, *d_b, *d_c;
  hipMalloc(&d_a, N * sizeof(float));
  hipMalloc(&d_b, N * sizeof(float));
  hipMalloc(&d_c, N * sizeof(float));

  hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);

  int numBlocks = N / 256;
  if (N % 256 != 0) {
    numBlocks++;
  }

  vecAdd<<<numBlocks, 256>>>(d_a, d_b, d_c);

  hipMemcpy(h_c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);

  for (int i = 0; i < N; i++) {
    printf("%f\n", h_c[i]);
  }

  free(h_a);
  free(h_b);
  free(h_c);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}