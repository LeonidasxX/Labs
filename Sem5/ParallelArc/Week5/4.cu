#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 10

__global__ void vecAdd(float *a, float *c) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < N) {
    c[id] = sinf(id);
  }
}

int main() {
  float *h_a = (float *)malloc(N * sizeof(float));
  float *h_b = (float *)malloc(N * sizeof(float));
  float *h_c = (float *)malloc(N * sizeof(float));

  printf("Enter radian array: ");
  for (int i = 0; i < N; i++) {
    scanf("%f", &h_a[i]);
  }

  float *d_a, *d_b, *d_c;
  hipMalloc(&d_a, N * sizeof(float));
  hipMalloc(&d_b, N * sizeof(float));
  hipMalloc(&d_c, N * sizeof(float));

  hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);

  vecAdd<<<N, N>>>(d_a, d_c);

  hipMemcpy(h_c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);

  printf("After Sin we get: \n");
  for (int i = 0; i < N; i++) {
    printf("%f\n", h_c[i]);
  }

  free(h_a);
  free(h_b);
  free(h_c);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}