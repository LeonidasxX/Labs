#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define width 1024

#define mask_width 3

__global__ void convolution(float *N, float *M, float *P) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < width) {
    float p = 0;
    for (int i = 0; i < mask_width; i++) {
      p += N[id + i - mask_width / 2] * M[i];
    }
    P[id] = p;
  }
}

int main() {
  float *h_N = (float *)malloc(width * sizeof(float));
  for (int i = 0; i < width; i++) {
    h_N[i] = i;
  }

  float *h_M = (float *)malloc(mask_width * sizeof(float));
  for (int i = 0; i < mask_width; i++) {
    h_M[i] = 1;
  }

  float *h_P = (float *)malloc(width * sizeof(float));

  float *d_N;
  hipMalloc(&d_N, width * sizeof(float));
  float *d_M;
  hipMalloc(&d_M, mask_width * sizeof(float));
  float *d_P;
  hipMalloc(&d_P, width * sizeof(float));

  hipMemcpy(d_N, h_N, width * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_M, h_M, mask_width * sizeof(float), hipMemcpyHostToDevice);

  convolution<<<width / 1024, 1024>>>(d_N, d_M, d_P);

  hipMemcpy(h_P, d_P, width * sizeof(float), hipMemcpyDeviceToHost);

  for (int i = 0; i < width; i++) {
    printf("%f\n", h_P[i]);
  }

  free(h_N);
  free(h_M);
  free(h_P);

  hipFree(d_N);
  hipFree(d_M);
  hipFree(d_P);

  return 0;
}