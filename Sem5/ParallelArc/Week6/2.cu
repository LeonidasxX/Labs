#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define N 1024

__global__ void CUDACopy(char* d_a, char* d_b, int len_a, int len_b) {
	int tid = threadIdx.x;
	int diff = len_a - tid;
	int start = len_b - (diff * (diff + 1)) / 2;
	for(int i=0; i<len_a-tid; i++)
		d_b[start+i] = d_a[i];
}


int main() {
	char a[N], b[N];
	char* d_a, *d_b;
	unsigned int size, len;
	
	printf("Enter a string: ");
	scanf(" %[^\n]s", a);
	
	len = strlen(a);
	len = (len * (len + 1)) / 2;
	size = len * sizeof(char);
	
	hipMalloc((void**) &d_a, strlen(a) * sizeof(char));
	hipMalloc((void**) &d_b, size);
	hipMemcpy(d_a, a, strlen(a) * sizeof(char), hipMemcpyHostToDevice);
	
	CUDACopy<<<1, strlen(a)>>>(d_a, d_b, strlen(a), len);

	hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost);
	b[len] = '\0';
	
	printf("Output String: %s\n", b);
	
	hipFree(d_a); hipFree(d_b);

	return 0;
}