#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define N 1024

__global__ void CUDACount(char* A, char* W, unsigned int l_W, unsigned int* d_count) {
	int tid = threadIdx.x;
	for(int i=0; i<l_W; i++) 
		if(A[tid+i] != W[i]) 
			return;
	atomicAdd(d_count, 1);
}

int main() {
	char A[N], B[N];
	char* d_A, *d_W;
	unsigned int count=0, *d_count, result;
	
	printf("Enter a string: ");
	scanf(" %[^\n]s", A);
	
	printf("Enter the word to be searched: ");
	scanf(" %[^\n]s", B);
	
	hipMalloc((void**) &d_A, strlen(A) * sizeof(char));
	hipMalloc((void**) &d_W, strlen(B) * sizeof(char));
	hipMalloc((void**) &d_count, sizeof(unsigned int));
	hipMemcpy(d_A, A, strlen(A) * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(d_W, B, strlen(B) * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(d_count, &count, sizeof(unsigned int), hipMemcpyHostToDevice);
	
	CUDACount<<<1, strlen(A)-strlen(B)+1>>>(d_A, d_W, strlen(B), d_count);

	hipMemcpy(&result, d_count, sizeof(unsigned int), hipMemcpyDeviceToHost);
	
	printf("Total occurrences of %s: %u\n", B, result);
	
	hipFree(d_A); hipFree(d_W); hipFree(d_count);

	return 0;
}