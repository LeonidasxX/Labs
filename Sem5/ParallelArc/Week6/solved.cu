#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include<string.h>
// #include "conio.h"
#define N 1024
__global__ void CUDACount(char* A, unsigned int *d_count){
    int i = threadIdx.x;
    if(A[i]=='a')
        atomicAdd(d_count,1);
}
int main() {
    char A[N];
    char *d_A;
    unsigned int *count=0,*d_count,*result;
    count = (unsigned int*)malloc(N*sizeof(unsigned int));
    result = (unsigned int*)malloc(N*sizeof(unsigned int));
    printf("Enter a string: ");
    scanf("%[^\n]s", A);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    hipMalloc((void**)&d_A, strlen(A)*sizeof(char));
    hipMalloc((void **)&d_count,sizeof(unsigned int));
    hipMemcpy(d_A, A, strlen(A)*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_count,count,sizeof(unsigned int),hipMemcpyHostToDevice);
    hipError_t error =hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA Error1: %s\n", hipGetErrorString(error));
    }
    CUDACount<<<1,strlen(A)>>>(d_A,d_count);
    error =hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA Error2: %s\n", hipGetErrorString(error));
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    hipMemcpy(result, d_count, sizeof(unsigned int), hipMemcpyDeviceToHost);
    printf("Total occurences of a=%u",*result);
    printf("Time Taken=%f",elapsedTime);
    hipFree(d_A);
    hipFree(d_count);
    printf("\n");
    // getch();
    return 0;
}